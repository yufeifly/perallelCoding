#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/gpu/gpu.hpp>

using namespace std;
using namespace cv;
using namespace cv::gpu;
int conv[] = {-1, -1, -1, -1, 9, -1, -1, -1, -1};
const int w = 16 + 3 - 1;

__global__ void conv_kernel(const PtrStepSz<uchar3> gpuin,PtrStep<uchar3> gpuout, int* __restrict__ mask)
{
    __shared__ unsigned char mx[w][w];
    __shared__ unsigned char my[w][w];
    __shared__ unsigned char mz[w][w];
    int rs = blockIdx.x * blockDim.x;
    int cs = blockIdx.y * blockDim.y;
    for (int i = 0; i < w; ++i)
    {
        for (int j = 0; j < w; ++j)
        {
            uchar3 vi = gpuin(rs + i,cs + j);
            mx[i][j] =  vi.x;
            my[i][j] =  vi.y;
            mz[i][j] =  vi.z;
        }
    }
    __syncthreads();
    uchar3 v;
    int vx = 0, vy = 0, vz = 0;
    int r = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int c = threadIdx.y + blockIdx.y * blockDim.y + 1;
    for (int i = 0; i < 3; ++i)
    {
        for (int j = 0; j < 3; ++j)
        {
            int ind = j+i*3;

            vx += mx[threadIdx.x + i][threadIdx.y + j] * mask[ind];
            vy += my[threadIdx.x + i][threadIdx.y + j] * mask[ind];
            vz += mz[threadIdx.x + i][threadIdx.y + j] * mask[ind];
        }
    }

	if(vx < 0)
		vx = 0;
	else if(vx > 255)
		vx = 255;
	if(vy < 0)
		vy = 0;
	else if(vy > 255)
		vy = 255;
	if(vz < 0)
		vz = 0;
	else if(vz > 255)
		vz = 255;
	v.x = vx;
	v.y = vy;
	v.z = vz;
    gpuout(r,c) = make_uchar3(v.x,v.y,v.z);
}

int main()
{
    cout << getCudaEnabledDeviceCount() << endl;
    DeviceInfo dev_inf(0);
    cout << dev_inf.isCompatible() << endl;
    setDevice(0);
    cout << dev_inf.multiProcessorCount() << endl;
    cout << dev_inf.totalMemory() << endl;
   
    Mat myMat = imread("../t.png");
    myMat.convertTo(myMat, CV_8U);
    Mat result;
    result.create(myMat.size(), myMat.type());
    GpuMat gpuin,gpuout;
    int *mask;
    gpuin.upload(myMat);
    gpuout.create(gpuin.size(), gpuin.type());
    dim3 block(16,16);
    dim3 grid((myMat.rows-2)/16,(myMat.cols-2)/16);
    hipMalloc((void **)&mask, sizeof(int)*9);
    hipMemcpy(mask, conv, sizeof(int)*9, hipMemcpyHostToDevice);

	double ta = (double)getTickCount();

    conv_kernel<<<grid,block>>>(gpuin, gpuout, mask);

	ta = ((double)getTickCount() - ta)/getTickFrequency();

    gpuout.download(result);
    result.row(0).setTo(Scalar(0));             // 上边界
    result.row(result.rows-1).setTo(Scalar(0)); // 下边界
    result.col(0).setTo(Scalar(0));             // 左边界
    result.col(result.cols-1).setTo(Scalar(0)); // 右边界
   
    cout << "times passed in seconds: " << ta << endl;
    hipFree(mask);
    imwrite("t3.png",result);

    return 0;
}