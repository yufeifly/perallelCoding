#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/gpu/gpu.hpp>

using namespace std;
using namespace cv;
using namespace cv::gpu;
int conv[] = {-1, -1, -1, -1, 9, -1, -1, -1, -1};

__global__ void conv_kernel(const PtrStepSz<uchar3> gpuin,PtrStep<uchar3> gpuout, int *mask)
{
    uchar3 v;
    int vx = 0, vy = 0, vz = 0;
    int r = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int c = threadIdx.y + blockIdx.y * blockDim.y + 1;
    for (int i = -1; i < 2; ++i)
    {
        for (int j = -1; j < 2; ++j)
        {
            int ind = 4+j+i*3;

            uchar3 vi = gpuin(r+i,c+j);
            vx += vi.x * mask[ind];
            vy += vi.y * mask[ind];
            vz += vi.z * mask[ind];
        }
    }

	if(vx < 0)
		vx = 0;
	else if(vx > 255)
		vx = 255;
	if(vy < 0)
		vy = 0;
	else if(vy > 255)
		vy = 255;
	if(vz < 0)
		vz = 0;
	else if(vz > 255)
		vz = 255;
	v.x = vx;
	v.y = vy;
	v.z = vz;
    gpuout(r,c) = make_uchar3(v.x,v.y,v.z);
}

int main()
{
    cout << getCudaEnabledDeviceCount() << endl;
    DeviceInfo dev_inf(0);
    cout << dev_inf.isCompatible() << endl;
    setDevice(0);
    cout << dev_inf.multiProcessorCount() << endl;
    cout << dev_inf.totalMemory() << endl;

    Mat myMat = imread("../t.png");
    myMat.convertTo(myMat, CV_8U);
    Mat result;
    result.create(myMat.size(), myMat.type());
    GpuMat gpuin,gpuout;
    int *mask;
    gpuin.upload(myMat);
    gpuout.create(gpuin.size(), gpuin.type());
	
    dim3 block(16,16);
    dim3 grid((myMat.rows-2)/16,(myMat.cols-2)/16);
    hipMalloc((void **)&mask, sizeof(int)*9);
    hipMemcpy(mask, conv, sizeof(int)*9, hipMemcpyHostToDevice);

    double ta = (double)getTickCount();

    conv_kernel<<<grid,block>>>(gpuin, gpuout, mask);

    ta = ((double)getTickCount() - ta)/getTickFrequency();
    gpuout.download(result);
    result.row(0).setTo(Scalar(0));             // 上边界
    result.row(result.rows-1).setTo(Scalar(0)); // 下边界
    result.col(0).setTo(Scalar(0));             // 左边界
    result.col(result.cols-1).setTo(Scalar(0)); // 右边界
    cout << "times passed in seconds: " << ta << endl;
    hipFree(mask);
    imwrite("t2.png",result);

    return 0;
}